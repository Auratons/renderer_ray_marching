#include "hip/hip_runtime.h"
#include <iostream>

#include "utils.h"
#include "common.h"
#include "ray_marching.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <glm/glm.hpp>
#include <hip/hip_vector_types.h>
#include <kdtree/kdtree_flann.h>
#include <stb_image_write.h>
#include <thrust/device_vector.h>

__device__ glm::mat4     VIEW;
__device__ float         FOV_RADIANS;
__device__ const float4 *VERTICES;
__device__ const float4 *COLORS;
__device__ const float  *RADII;
__device__ size_t        POINTCLOUD_SIZE;

__device__ float *FRUSTRUM_EDGE_PTS_WORLD_TMP;
__device__ size_t FRUSTRUM_VERTICES_CNT;
__device__ size_t *FRUSTRUM_VERTICES_IDX;

GLuint TEXTURE_HANDLE;

PointcloudRayMarcher *PointcloudRayMarcher::instance = nullptr;

surface<void, cudaSurfaceType2D> surfaceWrite; // NOLINT(cert-err58-cpp)
hipGraphicsResource_t           cuda_image_resource_handle;
hipArray_t                      cuda_image;


struct RayHit {
    float distance = 1.0f / 0.0f;  // MAX_FLOAT
    size_t index = 0;
};

__device__ RayHit distance_function(float3 pos);
__device__ long int ray_march(const float3 &ray_origin, const float3 &ray_dir);

template<typename T>
__device__ float3 make_float3(const T &v) {
  return make_float3(v.x, v.y, v.z);
}


__global__ void render()
{
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x >= SCREEN_WIDTH || y >= SCREEN_HEIGHT) {
    return;
  }
  auto resolution = make_float2(SCREEN_WIDTH, SCREEN_HEIGHT);
  auto coordinates = make_float2((float)x, (float)y);
  auto uv = (2.0f * coordinates - resolution);
  // In world coords
  auto camera_rot = glm::transpose(glm::mat3(VIEW));
  auto camera_pos = - camera_rot * glm::vec3(VIEW[3]);
  auto pane_dist = SCREEN_WIDTH / (2.0f * tanf(0.5f * FOV_RADIANS));
  auto ro = make_float3(camera_pos);
  auto rd = make_float3(glm::normalize(camera_rot * glm::vec3(uv.x, uv.y, -pane_dist)));
  auto color_index = ray_march(ro, rd);
  // Great life-saving trick for debugging purposes when not writing to the whole picture. Leaving as a memento.
  // auto finalColor = make_float4(x / (SCREEN_WIDTH-1), y / (SCREEN_HEIGHT-1), 1, 1);
  auto finalColor = BACKGROUND_COLOR;
  if (color_index >= 0) {
    finalColor = COLORS[color_index];
  }
#pragma diag_suppress 1215  // Deprecated symbol
  surf2Dwrite(finalColor, surfaceWrite, x * sizeof(float4), y);
#pragma diag_default 1215  // Deprecated symbol get back default behavior
  __syncthreads();
}

void PointcloudRayMarcher::render_to_texture(
  const glm::mat4 &view,
  float fov_radians) {
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(VIEW), &view, sizeof(view)) );
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(FOV_RADIANS), &fov_radians, sizeof(fov_radians)) );

  // Generate homogeneous point for a frustrum-edge-lying point
  auto v = [fov_radians] (float x, float y){
    auto x_factor = glm::tan(fov_radians / 2);
    return glm::vec3(x, y, -2 * x_factor / SCREEN_WIDTH);  // Camera looking to -z
  };
  // Due to computational cost, we're performing the test when moved to world origin.
  auto cam_to_world_rot = glm::transpose(glm::mat3(view));
  frustrum_edge_pts_world_tmp[0] = glm::vec4(cam_to_world_rot[2] + v(-1, -1), 1.0f);
  frustrum_edge_pts_world_tmp[1] = glm::vec4(cam_to_world_rot[2] + v(1, -1), 1.0f);
  frustrum_edge_pts_world_tmp[2] = glm::vec4(cam_to_world_rot[2] + v(1, 1), 1.0f);
  frustrum_edge_pts_world_tmp[3] = glm::vec4(cam_to_world_rot[2] + v(-1, 1), 1.0f);

  frustrum_pcd_size = thrust::copy_if(
    thrust::counting_iterator<size_t>(0), thrust::counting_iterator<size_t>(vertices.size()),
    vertices.begin(),
    frustrum_vertices_idx.begin(),
    [cam_pos = -view[3], cam_dir = glm::vec4(cam_to_world_rot[2], 0)] __device__ (const glm::vec4 &vertex){
      auto pt = vertex - cam_pos;
      auto in = true;
      // Frustrum
      for (int i = 0; i < 4; ++i) {
        // Vector pairing {{0, 1}, {1, 2}, {2, 3}, {3, 0}}
        auto v2 = FRUSTRUM_EDGE_PTS_WORLD_TMP + 4 * (i);
        auto v1 = FRUSTRUM_EDGE_PTS_WORLD_TMP + 4 * ((i + 1) % 4);
        // Plane through origin, (v2 x v1) . pt
        in &= (((*(v2+1) * *(v1+2) - *(v2+2) * *(v1+1)) * pt.x +
                (*(v2+2) * *v1     - *v2     * *(v1+2)) * pt.y +
                (*v2     * *(v1+1) - *(v2+1) * *v1)     * pt.z) > 0);
      }
      pt -= cam_dir * ZNEAR;
      in &= (pt.x * cam_dir.x + pt.y * cam_dir.y + pt.z * cam_dir.z > 0);
      pt -= cam_dir * (ZFAR - ZNEAR);
      in &= (pt.x * - cam_dir.x + pt.y * - cam_dir.y + pt.z * - cam_dir.z > 0);
      return in;
    }
  ) - frustrum_vertices_idx.begin();
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(FRUSTRUM_VERTICES_CNT), &frustrum_pcd_size, sizeof(frustrum_pcd_size)) );

  CHECK_ERROR_CUDA(
    hipGraphicsGLRegisterImage(
      &cuda_image_resource_handle,
      TEXTURE_HANDLE,
      GL_TEXTURE_2D,
      hipGraphicsRegisterFlagsSurfaceLoadStore
    )
  );
  CHECK_ERROR_CUDA( hipGraphicsMapResources(1, &cuda_image_resource_handle) );
  CHECK_ERROR_CUDA( hipGraphicsSubResourceGetMappedArray(&cuda_image, cuda_image_resource_handle, 0, 0) );
  CHECK_ERROR_CUDA( cudaBindSurfaceToArray(surfaceWrite, cuda_image) );
  dim3 block_dim(32, 32, 1);
  dim3 grid_dim(ceil(SCREEN_WIDTH / block_dim.x), ceil(SCREEN_HEIGHT / block_dim.y), 1);
  render<<< grid_dim, block_dim >>>();
  CHECK_ERROR_CUDA();
  CHECK_ERROR_CUDA( hipGraphicsUnmapResources(1, &cuda_image_resource_handle) );
  CHECK_ERROR_CUDA( hipGraphicsUnregisterResource(cuda_image_resource_handle) );
}

void PointcloudRayMarcher::save_png(const std::string &filename) {
  auto raw_data = texture.get_texture_data<float4>();
  auto png = std::vector<unsigned char>(4 * SCREEN_WIDTH * SCREEN_HEIGHT);  // 4=RGBA
  auto begin = (const float*)raw_data.data();
  auto end = (const float*)(raw_data.data() + raw_data.size());
  std::transform(begin, end, png.begin(), [](const float &val){ return (unsigned char)(val * 255.0f); });
  // OpenGL expects the 0.0 coordinate on the y-axis to be on the bottom side of the image, but images usually
  // have 0.0 at the top of the y-axis. For now, this unifies output with the visualisation on the screen.
  stbi_flip_vertically_on_write(true);
  stbi_write_png(filename.c_str(), SCREEN_WIDTH, SCREEN_HEIGHT, 4, png.data(), 4 * SCREEN_WIDTH);  // 4=RGBA
}

/*
 * Not thread safe.
 */
PointcloudRayMarcher *PointcloudRayMarcher::get_instance(
  const thrust::device_vector<glm::vec4> &vertices,
  const thrust::device_vector<glm::vec4> &colors,
  const thrust::device_vector<float> &radii,
  const Texture2D &texture) {
  TEXTURE_HANDLE = texture.get_id();
  if(instance == nullptr) {
    instance = new PointcloudRayMarcher(vertices, colors, radii, texture);
  }
  return instance;
}

PointcloudRayMarcher::PointcloudRayMarcher(
  const thrust::device_vector<glm::vec4> &vertices,
  const thrust::device_vector<glm::vec4> &colors,
  const thrust::device_vector<float> &radii,
  const Texture2D &texture) : vertices(vertices), colors(colors), radii(radii), texture(texture) {
  auto ptr = reinterpret_cast<const float4 *>(vertices.data().get());
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(VERTICES), &ptr, sizeof(ptr)) );
  ptr = reinterpret_cast<const float4 *>(colors.data().get());
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(COLORS), &ptr, sizeof(ptr)) );
  auto ptr_f = radii.data().get();
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(RADII), &ptr_f, sizeof(ptr_f)) );
  auto pointcloud_size = vertices.size();
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(POINTCLOUD_SIZE), &pointcloud_size, sizeof(pointcloud_size)) );
  auto ptr_v = reinterpret_cast<float *>(frustrum_edge_pts_world_tmp.data().get());
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(FRUSTRUM_EDGE_PTS_WORLD_TMP), &ptr_v, sizeof(ptr_v)) );
  frustrum_vertices_idx.resize(pointcloud_size);
  auto ptr_s = frustrum_vertices_idx.data().get();
  CHECK_ERROR_CUDA( hipMemcpyToSymbol(HIP_SYMBOL(FRUSTRUM_VERTICES_IDX), &ptr_s, sizeof(ptr_s)) );
}

__device__ long int ray_march(const float3 &ray_origin, const float3 &ray_dir) {
  float total_distance_travelled = 0.0f;
  float3 current_position;
  RayHit res;

  for (int i = 0; i < MAX_STEPS; ++i) {
    current_position = ray_origin + ray_dir * total_distance_travelled;
    res = distance_function(current_position);
    total_distance_travelled += res.distance;
    if (res.distance < MIN_DIST) {
      return (long int)res.index;
    }
    if (total_distance_travelled > ZFAR)
      break;
  }

  return -1;
}

__device__ RayHit distance_function(float3 pos) {
  float dist;
  RayHit hit;
  size_t index;
  for (size_t i = 0; i < FRUSTRUM_VERTICES_CNT; ++i) {
    index = FRUSTRUM_VERTICES_IDX[i];
    dist = length(make_float3(VERTICES[index]) - pos) - RADII[index];
    if (dist < hit.distance) {
      hit.distance = dist;
      hit.index = index;
    }
  }
  return hit;
}
